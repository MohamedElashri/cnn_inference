// File: src/common/cuda_common.cu

#include "common/cuda_common.cuh"

namespace unet {

// Define global handles
hipdnnHandle_t cudnn_handle;
hipblasHandle_t cublas_handle;

} // namespace unet